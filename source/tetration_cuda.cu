#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

#include "builder/builder.c"
#include "tetration.h"






/**
 *
 *  Main program
 *
 */

#ifndef M_PI_2
#define M_PI_2 1.5707963267948966
#endif

#ifndef M_SQRT2
#define M_SQRT2 1.4142135623730951
#endif

__device__ float modulus(float a, float b) {
    return sqrt(a * a + b * b);
}

__global__ void computeFractal(float* results, sysinfo_t sys_fractal, float width, float height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= width || idy >= height) return;

    float i = sys_fractal.x1 + idx * sys_fractal.pixel_step;
    float j = sys_fractal.y1 + idy * sys_fractal.pixel_step;

    float x = i, y = j, m, p, a, b;
    float angle = 0.0, maxAngle = 0.0;

    m = 0.5 * log(i * i + j * j);
    p = atan2(j, i);
    
    for (int r = 0; r < sys_fractal.iterations && modulus(x, y) < DBL_MAX; r++) {
        
        a = exp(x * m - p * y);
        b = y * m + p * x;
        x = a * cos(b);
        y = a * sin(b);

        angle = fabs(atan(y / x));
        //angle = fabs(atan2(x,y));

        if (r > 0 && angle > maxAngle)
            maxAngle = angle;
    }

    results[idy * (int)width + idx] = maxAngle / (M_PI_2);
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    
    userinfo_t usr_fractal = {0};
    usr_fractal.width = USR_WIDTH * USR_AAF;
    usr_fractal.height = USR_HEIGHT * USR_AAF;
    usr_fractal.center_x = USR_CENTER_X;
    usr_fractal.center_y = USR_CENTER_Y;
    usr_fractal.log_zoom = USR_LOGZOOM;
    usr_fractal.std_diagonal = 10.0 * M_SQRT2;

    float diagonal = usr_fractal.std_diagonal / pow(2.0, usr_fractal.log_zoom);
    float diag_to_width = diagonal / sqrt(1.0 + pow(usr_fractal.height / usr_fractal.width, 2));
    float half_width = diag_to_width * 0.5;
    float half_height = diag_to_width * (usr_fractal.height / usr_fractal.width) * 0.5;

    sysinfo_t sys_fractal = {0};
    sys_fractal.x1 = usr_fractal.center_x - half_width;
    sys_fractal.x2 = usr_fractal.center_x + half_width;
    sys_fractal.y1 = usr_fractal.center_y - half_height;
    sys_fractal.y2 = usr_fractal.center_y + half_height;
    sys_fractal.pixel_step = diag_to_width / usr_fractal.width;
    sys_fractal.iterations = ITERATIONS;

    int total_pixels = (int)(usr_fractal.width * usr_fractal.height);
    
    float* results = (float*)malloc(total_pixels * sizeof(float));
    if (results == NULL) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }
    
    
    
    // Nvidia CUDA specific commands

    float* d_results;
    checkCudaError(hipMalloc(&d_results, total_pixels * sizeof(float)), "Allocating device memory");

    int N = 32;  // Choose a reasonable block size
    dim3 blockDim(N, N);
    dim3 gridDim((int)ceil(usr_fractal.width / (float)N), (int)ceil(usr_fractal.height / (float)N));

    computeFractal<<<gridDim, blockDim>>>(d_results, sys_fractal, usr_fractal.width, usr_fractal.height);
    checkCudaError(hipGetLastError(), "Launching kernel");
    checkCudaError(hipDeviceSynchronize(), "Synchronizing after kernel");

    checkCudaError(hipMemcpy(results, d_results, total_pixels * sizeof(float), hipMemcpyDeviceToHost), "Copying results to host");

    hipFree(d_results);
    
    
    
    // Back to normal C
    
    builder_DataBuild(usr_fractal, results, BUILD_AAF, BUILD_COLORMAP);
    system("start " FILE_BITMAP);
    
    FILE* fp = fopen(FILE_RAWOUT, "wb");
    if (fp == NULL) {
        fprintf(stderr, "Failed to open file for writing.\n");
        return EXIT_FAILURE;
    }
    
    fwrite(&usr_fractal, sizeof(userinfo_t), 1, fp);
    fwrite(results, sizeof(float), total_pixels, fp);
    fclose(fp);
    free(results);

    return 0;
}
